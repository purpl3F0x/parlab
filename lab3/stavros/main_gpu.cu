#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h> /* strtok() */
#include <sys/stat.h>
#include <sys/types.h> /* open() */
#include <unistd.h>    /* getopt() */

int _debug;
#include "alloc.h"
#include "error.h"
#include "kmeans.h"


/// This is the validation eps that will be used for result comparisson.
/// If set too low implementations with very divergent double operation order might lead to errors!
/// Always check error difference in these cases and adjust eps accordingly!
#ifdef VALIDATE
double validation_eps = 1e-2;
#endif

static void usage(char* argv0) {
    char* help = "Usage: %s [switches]\n"
                 "       -c num_clusters    : number of clusters (must be > 1)\n"
                 "       -s size            : size of examined dataset\n"
                 "       -n num_coords      : number of coordinates\n"
                 "       -t threshold       : threshold value (default : 0.001)\n"
                 "       -l loop_threshold  : iterations threshold (default : 10)\n"
                 "       -d                 : enable debug mode\n"
                 "       -h                 : print this help information\n"
                 "GPU extras:\n"
                 "       -b                 : blocksize\n";
    fprintf(stderr, help, argv0);
    exit(-1);
}

void DisplayHeader() {
    const int kb = 1024;
    const int mb = kb * kb;
    printf("NBody.GPU\n=========\n\n");

    printf("CUDA version:   v%d\n", CUDART_VERSION);
    printf("Thrust version: v%d.%d\n\n", THRUST_MAJOR_VERSION, THRUST_MINOR_VERSION);

    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Devices: \n\n");

    for (int i = 0; i < devCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%d: %s: %d.%d\n", i, props.name, props.major, props.minor);
        printf("  Global memory:   %ldmb\n", props.totalGlobalMem / mb);
        printf("  Shared memory:   %dkb\n", props.sharedMemPerBlock / kb);
        printf("  Constant memory: %dkb\n", props.totalConstMem / kb);
        printf("  Block registers: %d\n\n", props.regsPerBlock);

        printf("  Warp size:         %d\n", props.warpSize);
        printf("  Threads per block: %d\n", props.maxThreadsPerBlock);
        printf("  Max block dimensions: [ %d, %d, %d ]\n",
               props.maxThreadsDim[0],
               props.maxThreadsDim[1],
               props.maxThreadsDim[2]);
        printf("  Max grid dimensions:  [ %d, %d, %d ]\n",
               props.maxGridSize[0],
               props.maxGridSize[1],
               props.maxGridSize[2]);
        printf("\n");
    }
}

int main(int argc, char** argv) {
    long         i, j, opt;
    extern char* optarg;
    extern int   optind;

    int block_size = 0;

    long    numClusters = 0, numCoords = 0, numObjs = 0;
    int*    membership; // [numObjs]
    double* objects;    // [numObjs * numCoords] data  objects
    double* clusters;   // [numClusters * numCoords] cluster center
    double  dataset_size = 0, threshold;
    long    loop_threshold;
    double  io_timing_read;

    /* some default values */
    _debug         = 0;
    threshold      = 0.001;
    loop_threshold = 10;
    numClusters    = 0;

    printf(
      "\n~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n");

    while ((opt = getopt(argc, argv, "b:n:t:l:c:s:dh")) != EOF) {
        switch (opt) {
            case 'b':
                block_size = atol(optarg);
                break;
            case 'c':
                numClusters = atol(optarg);
                break;
            case 't':
                threshold = atof(optarg);
                break;
            case 'l':
                loop_threshold = atol(optarg);
                break;
            case 's':
                dataset_size = atof(optarg);
                break;
            case 'n':
                numCoords = atol(optarg);
                break;
            case 'd':
                _debug = 1;
                break;
            case 'h':
            default:
                usage(argv[0]);
                break;
        }
    }
    if (!block_size)
        error("block_size not provided for GPU version, terminating\n");

    if (numClusters <= 1)
        usage(argv[0]);

    numObjs = (dataset_size * 1024 * 1024) / (numCoords * sizeof(double));

    if (numObjs < numClusters) {
        printf("Error: number of clusters must be larger than the number of data points to be "
               "clustered.\n");
        return 1;
    }

    DisplayHeader();

    puts("\n\n");

    printf("dataset_size = %.2f MB    numObjs = %ld    numCoords = %ld    numClusters = %ld, "
           "block_size = %d\n",
           dataset_size,
           numObjs,
           numCoords,
           numClusters,
           block_size);


    objects = dataset_generation(numObjs, numCoords);

    // Allocate space for clusters (coordinates of cluster centers)
    clusters = (double*)malloc(numClusters * numCoords * sizeof(double));

#ifdef VALIDATE
    // Allocate space for validation clusters (coordinates of cluster centers)
    double* validation_clusters = (double*)malloc(numClusters * numCoords * sizeof(double));
#endif

    // The first numClusters elements are selected as initial centers
    for (i = 0; i < numClusters; i++)
        for (j = 0; j < numCoords; j++) {
            clusters[i * numCoords + j] = objects[i * numCoords + j];
#ifdef VALIDATE
            validation_clusters[i * numCoords + j] = clusters[i * numCoords + j];
#endif
        }
    // check initial cluster centers for repetition
    if (check_repeated_clusters(numClusters, numCoords, clusters) == 0) {
        printf(
          "Error: some initial clusters are repeated. Please select distinct initial centers\n");
        return 1;
    }


    // printf("Initial cluster centers:\n");
    // for (i=0; i<numClusters; i++) {
    //     printf("clusters[%ld] =",i);
    //     for (j=0; j<numCoords; j++)
    //         printf(" %6.2f", clusters[i*numCoords + j]);
    //     printf("\n");
    // }


    // membership: the cluster id for each data object
    membership = (int*)malloc(numObjs * sizeof(int));

#ifdef VALIDATE
    // Perform validation run
    kmeans(objects,
           numCoords,
           numObjs,
           numClusters,
           threshold,
           loop_threshold,
           membership,
           validation_clusters);
#endif
    // start the core computation
    printf("\n");
    kmeans_gpu(objects,
               numCoords,
               numObjs,
               numClusters,
               threshold,
               loop_threshold,
               membership,
               clusters,
               block_size);
    printf("\n");


    // printf("Final cluster centers:\n");
    // for (i=0; i<numClusters; i++) {
    //     printf("clusters[%ld] = ",i);
    //    for (j=0; j<numCoords; j++)
    //         printf("%6.2f ", clusters[i*numCoords + j]);
    //     printf("\n");
    // }

#ifdef VALIDATE
    printf("Performing validation....");
    int ik, checked[numClusters][numCoords];
    for (i = 0; i < numClusters; i++)
        for (j = 0; j < numCoords; j++)
            checked[i][j] = 0;
    for (i = 0; i < numClusters; i++)
        for (j = 0; j < numCoords; j++)
            if (!checked[i][j]) {
                for (ik = 0; ik < numClusters; ik++)
                    if (!checked[i][j]) {
                        if (abs((validation_clusters[i * numCoords + j] -
                                 clusters[ik * numCoords + j]) /
                                validation_clusters[i * numCoords + j]) < validation_eps) {
                            checked[i][j] = 1;
                            break;
                        }
                    }
                if (!checked[i][j])
                    error("Validation failed: cluster[%d][%d]: %lf instead of %lf\n",
                          i,
                          j,
                          clusters[i * numCoords + j],
                          validation_clusters[i * numCoords + j]);
                // else printf("Validation ok: cluster[%d][%d]: %lf instead of %lf\n", i, j,
                // clusters[ik*numCoords + j],  validation_clusters[i*numCoords + j]);
            }
    printf("PASSED!\n");
    free(validation_clusters);
#endif
    free(objects);
    free(membership);
    free(clusters);

    return 0;
}
