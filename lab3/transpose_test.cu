#include "transpose.cuh"

#include <sys/time.h>

#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <thread>

double wtime(void) {
    double          now_time;
    struct timeval  etstart;
    struct timezone tzp;

    if (gettimeofday(&etstart, &tzp) == -1)
        perror("Error: calling gettimeofday() not successful.\n");

    now_time = ((double)etstart.tv_sec) +             // in seconds
               ((double)etstart.tv_usec) / 1000000.0; // in microseconds
    return now_time;
}

struct GpuTimer {
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start() {
        hipDeviceSynchronize();
        hipEventRecord(start, 0);
    }

    void Stop() {
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
    }

    float Elapsed() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

int main(int argc, char** argv) {
    const size_t M    = atoi(argv[1]);
    const size_t N    = atoi(argv[2]);
    const size_t size = M * N * sizeof(double);

    double* data     = (double*)malloc(size);
    double* data_cpu = (double*)malloc(size);

    GpuTimer timer;

    for (size_t i = 0; i < M; i++) {
        for (size_t j = 0; j < N; j++) {
            data[i * N + j]     = i * N + j;
            data_cpu[i * N + j] = i * N + j;
        }
    }

    // GPU Transpose
    double* d_data;

    timer.Start();
    hipMalloc(&d_data, size);
    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
    transpose_inplace(d_data, M, N);
    hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
    timer.Stop();

    // CPU Transpose
    auto start = wtime();
    for (size_t i = 0; i < M; i++) {
        for (size_t j = 0; j < i; j++) {
            auto tmp            = data_cpu[i * N + j];
            data_cpu[i * N + j] = data_cpu[j * M + i];
            data_cpu[j * M + i] = tmp;
            std::this_thread::sleep_for(std::chrono::nanoseconds(1));
        }
    }
    auto stop = wtime();


    printf("Transpose test passed\n");
    printf("GPU Time: %.3f ms\n", timer.Elapsed());
    printf("CPU Time: %.3f ms\n", (stop - start) * 1000);

    // Compare
    for (size_t i = 0; i < M; i++) {
        for (size_t j = 0; j < N; j++) {
            if (data[i * N + j] != data_cpu[i * N + j]) {
                printf(
                  "Mismatch at (%d, %d): %f != %f\n", i, j, data[i * N + j], data_cpu[i * N + j]);
                return 1;
            }
        }
    }

    // // print gpu result
    // for (size_t i = 0; i < M; i++) {
    //     for (size_t j = 0; j < N; j++) {
    //         printf("%.0f ", data[i * N + j]);
    //     }
    //     printf("\n");
    // }

    free(data);
    free(data_cpu);
    hipFree(d_data);

    return 0;
}