#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ __forceinline__ int get_tid() {
  return blockDim.x * blockIdx.x + threadIdx.x;
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                               int numObjs,
                               int numClusters,
                               double *objects,     // [numCoords][numObjs]
                               double *clusters,    // [numCoords][numClusters]
                               int objectId,
                               int clusterId) {
  int i;
  double diff;
  double ans = 0.0;

  for(i = 0; i < numCoords; ++i) {
    diff = objects[i * numObjs + objectId] - clusters[i * numClusters + clusterId];
    ans += diff * diff;
  }

  return (ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *deviceobjects,           //  [numCoords][numObjs]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *devicenewClusterSize,
                          double *devicenewClusters,
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmemClusters[];

  for(int i = threadIdx.x; i < numClusters; i += blockDim.x) {
    for(int j = 0; j < numCoords; ++j) {
      shmemClusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
    }
  }
  __syncthreads();

  const int tid = get_tid();

  if (tid < numObjs) {
    int index, i;
    double dist, min_dist;

    index = 0;

    min_dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, 0);

    for (i = 1; i < numClusters; i++) {
      dist = euclid_dist_2_transpose(numCoords, numObjs, numClusters, deviceobjects, shmemClusters, tid, i);

      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (deviceMembership[tid] != index) {
      /* TODO: Maybe something is missing here... is this write safe? */
      atomicAdd(devdelta, 1.0);
    }

    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;

    atomicAdd(&devicenewClusterSize[index], 1);

    for(int j = 0; j < numCoords; ++j) {
      atomicAdd(&devicenewClusters[j * numClusters + index], deviceobjects[tid + j * numObjs]);    
    }
  }
}

__global__ static
void update_centroids(int numCoords,
                      int numClusters,
                      int *devicenewClusterSize,           //  [numClusters]
                      double *devicenewClusters,    //  [numCoords][numClusters]
                      double *deviceClusters)    //  [numCoords][numClusters])
{

  const int tid = get_tid();
  const int clusterId = tid % numClusters;

  if (tid < numCoords * numClusters) {
  
    if (devicenewClusterSize[clusterId] > 0)
      deviceClusters[tid] = devicenewClusters[tid] / devicenewClusterSize[clusterId];

    devicenewClusters[tid] = 0;
  }
  __syncthreads();
  devicenewClusterSize[clusterId] = 0;
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */
void kmeans_gpu(double *objects,      /* in: [numObjs][numCoords] */
                int numCoords,    /* no. features */
                int numObjs,      /* no. objects */
                int numClusters,  /* no. clusters */
                double threshold,    /* % objects change membership */
                long loop_threshold,   /* maximum number of iterations */
                int *membership,   /* out: [numObjs] */
                double *clusters,   /* out: [numClusters][numCoords] */
                int blockSize) {
  double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
  double timing_gpu, timing_cpu, timing_transfers, transfers_time = 0.0, cpu_time = 0.0, gpu_time = 0.0;
  double alloc_time, gpu_alloc_time, gpu_get_time;
  int loop_iterations = 0;
  int i, j, index, loop = 0;
  double delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */

  /* TODO: Copy me from transpose version*/
  double **dimObjects = (double **) calloc_2d(numCoords, numObjs, sizeof(double));
  double **dimClusters = (double **) calloc_2d(numCoords, numClusters, sizeof(double));
  double **newClusters = (double **) calloc_2d(numCoords, numClusters, sizeof(double));

  printf("\n|-----------Full-offload GPU Kmeans------------|\n\n");

  double *deviceObjects;
  double *deviceClusters, *devicenewClusters;
  int *deviceMembership;
  int *devicenewClusterSize; /* [numClusters]: no. objects assigned in each new cluster */

  for (i = 0; i < numObjs; ++i) {
    for (j = 0; j < numCoords; ++j) {
      dimObjects[j][i] = objects[i * numCoords + j];
    }
  }


  /* pick first numClusters elements of objects[] as initial cluster centers*/
  for (i = 0; i < numCoords; i++) {
    for (j = 0; j < numClusters; j++) {
      dimClusters[i][j] = dimObjects[i][j];
    }
  }

  /* initialize membership[] */
  for (i = 0; i < numObjs; i++) membership[i] = -1;

  timing = wtime() - timing;
  alloc_time = timing * 1000;
  printf("t_alloc: %lf ms\n\n", 1000 * timing);
  timing = wtime();
  const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize) ? blockSize : numObjs;
  const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock; /* TODO: Calculate Grid size, e.g. number of blocks. */

  /*	Define the shared memory needed per block.
      - BEWARE: We can overrun our shared memory here if there are too many
      clusters or too many coordinates!
      - This can lead to occupancy problems or even inability to run.
      - Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
  const unsigned int clusterBlockSharedDataSize = numClusters * numCoords * sizeof(double);

  hipDeviceProp_t deviceProp;
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipGetDeviceProperties(&deviceProp, deviceNum);

  if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
    error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
  }

  checkCuda(hipMalloc(&deviceObjects, numObjs * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&deviceClusters, numClusters * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&devicenewClusters, numClusters * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&devicenewClusterSize, numClusters * sizeof(int)));
  checkCuda(hipMalloc(&deviceMembership, numObjs * sizeof(int)));
  checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));

  timing = wtime() - timing;
  gpu_alloc_time = timing * 1000;
  printf("t_alloc_gpu: %lf ms\n\n", 1000 * timing);
  timing = wtime();

  checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
                       numObjs * numCoords * sizeof(double), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(deviceMembership, membership,
                       numObjs * sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                       numClusters * numCoords * sizeof(double), hipMemcpyHostToDevice));
  checkCuda(hipMemset(devicenewClusterSize, 0, numClusters * sizeof(int)));
  free(dimObjects[0]);

  timing = wtime() - timing;
  gpu_get_time = timing * 1000;
  printf("t_get_gpu: %lf ms\n\n", 1000 * timing);
  timing = wtime();

  do {
    timing_internal = wtime();
    checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));
    timing_gpu = wtime();
    //printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
    /* TODO: change invocation if extra parameters needed
    find_nearest_cluster
        <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
        (numCoords, numObjs, numClusters,
         deviceObjects, devicenewClusterSize, devicenewClusters, deviceClusters, deviceMembership, dev_delta_ptr);
    */
    find_nearest_cluster<<<numClusterBlocks,
                            numThreadsPerClusterBlock,
                            clusterBlockSharedDataSize>>>(numCoords,
                                                          numObjs,
                                                          numClusters,
                                                          deviceObjects,
                                                          deviceClusters,
                                                          devicenewClusterSize,
                                                          devicenewClusters,
                                                          deviceMembership,
                                                          dev_delta_ptr);

    hipDeviceSynchronize();
    checkLastCudaError();

    gpu_time += wtime() - timing_gpu;

    //printf("Kernels complete for itter %d, updating data in CPU\n", loop);

    timing_transfers = wtime();
    checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
    transfers_time += wtime() - timing_transfers;

    const unsigned int update_centroids_block_sz = (numCoords * numClusters > blockSize) ? blockSize : numCoords *
                                                                                                       numClusters;  /* TODO: can use different blocksize here if deemed better */
    const unsigned int update_centroids_dim_sz = (numCoords * numClusters + update_centroids_block_sz - 1)/update_centroids_block_sz; /* TODO: calculate dim for "update_centroids" */
    timing_gpu = wtime();
    
    update_centroids<<<update_centroids_dim_sz, update_centroids_block_sz, 0>>>
    (numCoords, numClusters, devicenewClusterSize, devicenewClusters, deviceClusters);

    hipDeviceSynchronize();
    checkLastCudaError();
    gpu_time += wtime() - timing_gpu;

    timing_cpu = wtime();
    delta /= numObjs;
    //printf("delta is %f - ", delta);
    loop++;
    //printf("completed loop %d\n", loop);
    cpu_time += wtime() - timing_cpu;

    timing_internal = wtime() - timing_internal;
    if (timing_internal < timer_min) timer_min = timing_internal;
    if (timing_internal > timer_max) timer_max = timing_internal;
  } while (delta > threshold && loop < loop_threshold);

  checkCuda(hipMemcpy(membership, deviceMembership,
                       numObjs * sizeof(int), hipMemcpyDeviceToHost));
  checkCuda(hipMemcpy(dimClusters[0], deviceClusters,
                       numClusters * numCoords * sizeof(double), hipMemcpyDeviceToHost));

  for (i = 0; i < numClusters; i++) {
    for (j = 0; j < numCoords; j++) {
      clusters[i * numCoords + j] = dimClusters[j][i];
    }
  }

  timing = wtime() - timing;
  printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\t"
         "-> t_cpu_avg = %lf ms\n\t-> t_gpu_avg = %lf ms\n\t-> t_transfers_avg = %lf ms\n\n|-------------------------------------------|\n",
         loop, 1000 * timing, 1000 * timing / loop, 1000 * timer_min, 1000 * timer_max,
         1000 * cpu_time / loop, 1000 * gpu_time / loop, 1000 * transfers_time / loop);

  char outfile_name[1024] = {0};
  sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv",
          numObjs * numCoords * sizeof(double) / (1024 * 1024), numCoords, numClusters);
  FILE *fp = fopen(outfile_name, "a+");
  if (!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name);
  // fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "All_GPU", blockSize, timing / loop, timer_min, timer_max);
  fprintf(fp,
          "%s, %d, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf\n",
          "All_GPU",
          blockSize,
          timing * 1000,
          timing / loop * 1000,
          timer_min * 1000,
          timer_max * 1000,
          cpu_time / loop * 1000,
          gpu_time / loop * 1000,
          transfers_time / loop * 1000,
          alloc_time,
          gpu_alloc_time,
          gpu_get_time);

  fclose(fp);

  checkCuda(hipFree(deviceObjects));
  checkCuda(hipFree(deviceClusters));
  checkCuda(hipFree(devicenewClusters));
  checkCuda(hipFree(devicenewClusterSize));
  checkCuda(hipFree(deviceMembership));

  return;
}

