#pragma once
#include <hip/hip_runtime.h>


template<unsigned int blockSize>
__device__ __forceinline__ void warpReduce(volatile double* sdata, unsigned int tid) {
    if (blockSize >= 64)
        sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32)
        sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16)
        sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8)
        sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4)
        sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2)
        sdata[tid] += sdata[tid + 1];
}

template<unsigned int blockSize>
__global__ void reduce6(double* g_idata, double* g_odata, unsigned int n) {
    extern __shared__ double sdata[];
    unsigned int             tid      = threadIdx.x;
    unsigned int             i        = blockIdx.x * (blockSize * 2) + tid;
    unsigned int             gridSize = blockSize * 2 * gridDim.x;
    sdata[tid]                        = 0;
    while (i < n) {
        sdata[tid] += g_idata[i] + g_idata[i + blockSize];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }
    if (tid < 32)
        warpReduce<blockSize>(sdata, tid);
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}